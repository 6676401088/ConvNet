#include "hip/hip_runtime.h"
#include "ActivationLayer.h"
#include "Utility.h"

double __device__ sigmoid(double x) {
	//can only be called from device
	return 1.0 / (1.0 + exp(-x));
}

double __device__ sigmoidPrime(double x) {
	x = sigmoid(x);
	return x * (1 - x);
}

double __device__ softplus(double x) {
	return log(1 + exp(x));
}

double __device__ softplusPrime(double x) {
	return sigmoid(x);
}
double __device__ ReLU(double x) {
	return x > 0 ? x : 0;
}
double __device__ ReLUPrime(double x) {
	return x > 0 ? 1 : 0;
}

double __device__ mytanh(double x) {
	//in order to enforce device function ptr.
	return tanh(x);
}

double __device__ tanhPrime(double x) {
	x = tanh(x);
	return 1 - x * x;
	//return x * (1-x);
}
void __global__ sigmoid(double* I, double* O){
	int i = threadIdx.x;
	O[i]  = 1.0 / (1.0 + exp(-I[i]));
}
void __global__ activate(double* I, double* O, dfun f) {
	//can be called from host
	int i = threadIdx.x;
	O[i] = f(I[i]);
}
void __global__ activate(double* I, double* O, dfun f, int lim) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i<lim)
		O[i] = f(I[i]);
}
void activate(Matrix& I, Matrix& O, dfun f) {

	int n_elem = I.size().wh;
	//hline();
	//namedPrint(I);

	if(n_elem < 1024){
		activate<<<1, n_elem>>>
					(I.d_data(), O.d_data(), f);
	}else{
		activate<<< (n_elem+255) / 256, 256>>>
					(I.d_data(), O.d_data(), f, n_elem);
	}
	//O.set_sync(false);
	//namedPrint(O);
	//TODO: potentially divide up to more threads?
}

__device__ dfun pf_sig = sigmoid;
__device__ dfun pf_sig_d = sigmoidPrime;
__device__ dfun pf_sp = softplus;
__device__ dfun pf_sp_d = softplusPrime;
__device__ dfun pf_relu = ReLU;
__device__ dfun pf_relu_d = ReLUPrime;
__device__ dfun pf_tanh = mytanh;
__device__ dfun pf_tanh_d = tanhPrime;

ActivationLayer::ActivationLayer(std::string _f) {
	for (auto& c : _f) {
		c = std::tolower(c);
	}

	if (_f == "sigmoid") {
		hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf_sig), sizeof(dfun));
		hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_sig_d), sizeof(dfun));
	} else if (_f == "softplus") {
		hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf_sp), sizeof(dfun));
		hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_sp_d), sizeof(dfun));
	} else if (_f == "relu") {
		hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf_relu), sizeof(dfun));
		hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_relu_d), sizeof(dfun));
	} else if (_f == "tanh") {
		hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf_tanh), sizeof(dfun));
		hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_tanh_d), sizeof(dfun));
	} else {
		throw "WRONG ACTIVATION FUNCTION!!";
	}

}

ActivationLayer::~ActivationLayer(){

}

void ActivationLayer::setup(Size& _s, int& _d) {
	s = _s;
	d = _d;

	for (int i = 0; i < d; ++i) {
		I.push_back(Matrix(s));
		G.push_back(Matrix(s));
		O.push_back(Matrix(s));
	}

}

std::vector<Matrix>& ActivationLayer::FF(std::vector<Matrix>& _I) {
	for (int i = 0; i < d; ++i) {
		_I[i].copyTo(I[i]);
		//namedPrint(I[i]);
		//sigmoid<<<1,s.wh>>>(I[i].d_data(),O[i].d_data());
		activate(I[i], O[i], f);
		O[i].set_sync(false); //O[i] is not synced anymore!
		//namedPrint(O[i]);

	}
	return O;
}

std::vector<Matrix>& ActivationLayer::BP(std::vector<Matrix>& _G) {
	Matrix tmp(s);
	for (int i = 0; i < d; ++i) {
		activate(I[i], tmp, f_d);
		G[i] = _G[i] % tmp;
		//or consider setting G[i].dat as destination of mul.
	}
	return G;
}


void ActivationLayer::update(){

}
