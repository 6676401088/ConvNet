#include "hip/hip_runtime.h"
#include "ActivationLayer.h"
#include "Utility.h"

double __device__ sigmoid(double x) {
	//can only be called from device
	return 1.0 / (1.0 + exp(-x));
}

double __device__ sigmoidPrime(double x) {
	x = sigmoid(x);
	return x * (1 - x);
}

double __device__ softplus(double x) {
	return log(1 + exp(x));
}

double __device__ softplusPrime(double x) {
	return sigmoid(x);
}
double __device__ ReLU(double x) {
	return x > 0 ? x : 0;
}
double __device__ ReLUPrime(double x) {
	return x > 0 ? 1 : 0;
}

double __device__ mytanh(double x) {
	//in order to enforce device function ptr.
	return tanh(x);
}

double __device__ tanhPrime(double x) {
	x = tanh(x);
	return 1 - x * x;
	//return x * (1-x);
}
void __global__ activate(double* I, double* O, dfun f) {
	//can be called from host
	int i = threadIdx.x;
	O[i] = f(I[i]);
}
void activate(Matrix& I, Matrix& O, dfun f) {
	int n_elem = I.size().wh;
	activate<<<1, n_elem>>>
			(I.d_data(), O.d_data(), f);
	//TODO: potentially divide up to more threads?
}

__device__ dfun pf_sig = sigmoid;
__device__ dfun pf_sig_d = sigmoidPrime;
__device__ dfun pf_sp = softplus;
__device__ dfun pf_sp_d = softplusPrime;
__device__ dfun pf_relu = ReLU;
__device__ dfun pf_relu_d = ReLUPrime;
__device__ dfun pf_tanh = mytanh;
__device__ dfun pf_tanh_d = tanhPrime;

ActivationLayer::ActivationLayer(std::string _f) {


	for (auto& c : _f) {
		c = std::tolower(c);
	}

	//hipMalloc(&f,sizeof(dfun));
	//hipMalloc(&f_d,sizeof(dfun));

	if (_f == "sigmoid") {
		hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf_sig), sizeof(dfun),0,hipMemcpyDeviceToHost);
		hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_sig_d), sizeof(dfun),0,hipMemcpyDeviceToHost);
		//pf = sigmoid;
		//pf_d = sigmoidPrime;
		//hipMemcpyFromSymbol(&h_f, HIP_SYMBOL(sigmoid), sizeof(dfun));
		//hipMemcpyFromSymbol(&h_f_d, HIP_SYMBOL(sigmoidPrime), sizeof(dfun));
		//f = sigmoid;
		//f_d = sigmoidPrime;
	} else if (_f == "softplus") {
		hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf_sp), sizeof(dfun),0,hipMemcpyDeviceToHost);
		hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_sp_d), sizeof(dfun),0,hipMemcpyDeviceToHost);

		//pf = softplus;
		//pf_d = softplusPrime;
		//hipMemcpyFromSymbol(&h_f, HIP_SYMBOL(softplus), sizeof(dfun));
		//hipMemcpyFromSymbol(&h_f_d, HIP_SYMBOL(softplusPrime), sizeof(dfun));

		//f = softplus;
		//f_d = softplusPrime;
	} else if (_f == "relu") {
		hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf_relu), sizeof(dfun),0,hipMemcpyDeviceToHost);
		hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_relu_d), sizeof(dfun),0,hipMemcpyDeviceToHost);

		//pf = ReLU;
		//pf_d = ReLUPrime;
		//hipMemcpyFromSymbol(&h_f, HIP_SYMBOL(ReLU), sizeof(dfun));
		//hipMemcpyFromSymbol(&h_f_d, HIP_SYMBOL(ReLUPrime), sizeof(dfun));
		//f = ReLU;
		//f_d = ReLUPrime;
	} else if (_f == "tanh") {
		hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf_tanh), sizeof(dfun),0,hipMemcpyDeviceToHost);
		hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_tanh_d), sizeof(dfun),0,hipMemcpyDeviceToHost);

		//pf = mytanh;
		//pf_d = tanhPrime;
		//hipMemcpyFromSymbol(&h_f, HIP_SYMBOL(mytanh), sizeof(dfun));
		//hipMemcpyFromSymbol(&h_f_d, HIP_SYMBOL(tanhPrime), sizeof(dfun));
		//f = mytanh;
		//f_d = tanhPrime;
	} else {
		throw "WRONG ACTIVATION FUNCTION!!";
	}
	//dfun h_f; // = (dfun*)malloc(sizeof(dfun));
	//dfun h_f_d;// = (dfun*)malloc(sizeof(dfun));
	//hipMemcpyFromSymbol(&f, HIP_SYMBOL(pf), sizeof(dfun),0,hipMemcpyDeviceToHost);
	//hipMemcpyFromSymbol(&f_d, HIP_SYMBOL(pf_d), sizeof(dfun),0,hipMemcpyDeviceToHost);

	//hipMemcpy(f,&h_f,sizeof(dfun),hipMemcpyHostToDevice);
	//hipMemcpy(f_d,&h_f_d,sizeof(dfun),hipMemcpyHostToDevice);

}

ActivationLayer::~ActivationLayer(){
	//TODO : find out if freeing is necessary (there's probably reference)
	//hipFree(f);
	//hipFree(f_d);
}

void ActivationLayer::setup(Size& _s, int& _d) {
	s = _s;
	d = _d;

	for (int i = 0; i < d; ++i) {
		I.push_back(Matrix(s));
		G.push_back(Matrix(s));
		O.push_back(Matrix(s));
	}

}

std::vector<Matrix>& ActivationLayer::FF(std::vector<Matrix>& _I) {
	for (int i = 0; i < d; ++i) {
		_I[i].copyTo(I[i]);
		activate(I[i], O[i], f);
	}
	return O;
}

std::vector<Matrix>& ActivationLayer::BP(std::vector<Matrix>& _G) {
	Matrix tmp(s);
	for (int i = 0; i < d; ++i) {
		activate(I[i], tmp, f_d);
		G[i] = _G[i] % tmp;
		//or consider setting G[i].dat as destination of mul.
	}
	return G;
}


void ActivationLayer::update(){

}
