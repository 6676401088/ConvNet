#include "DropoutLayer.h"

bool DropoutLayer::enabled = true;

DropoutLayer::DropoutLayer(double p):p(p){

}

DropoutLayer::~DropoutLayer(){
	delete[] streams;
}
void DropoutLayer::setup(Size& _s, int& _d) {
	s = _s;
	d = _d;

	streams = new hipStream_t[d];
	for (int i = 0; i < d; ++i) {
		G.push_back(Matrix(s));
		O.push_back(Matrix(s));
		Mask.push_back(Matrix(s));
		hipStreamCreate(&streams[i]);
	}
}

std::vector<Matrix>& DropoutLayer::FF(std::vector<Matrix>& _I) {
	if(enabled){
		for (int i = 0; i < d; ++i) {
				//_I[i].copyTo(I[i]);
				Mask[i].randu(0.0,1.0);
				Mask[i] = (Mask[i] < p); //binary threshold
				O[i] = _I[i] % Mask[i];
			}
		return O;
	}else{
		return _I;
	}
}

std::vector<Matrix>& DropoutLayer::BP(std::vector<Matrix>& _G) {
	if(enabled){
		for (int i = 0; i < d; ++i) {
			G[i] = _G[i] % Mask[i];
			G[i] /= p;
		}
		return G;
	}else{
		return _G;
	}
}


void DropoutLayer::update(){

}

void DropoutLayer::enable(bool d){
	enabled = d;
}
