#include "FlattenLayer.h"

FlattenLayer::FlattenLayer() {
	d_in=0;
}

FlattenLayer::~FlattenLayer() {

}

void FlattenLayer::setup(Size& s, int& d) {
	d_in = d;
	s_in = s;
	s_out = Size(1, d * s.wh);

	O.push_back(Matrix(s_out));
	for (int i = 0; i < d_in; ++i) {
		I.push_back(Matrix(s_in)); //preallocating
		G.push_back(Matrix(s_in));
	}

	s = s_out;
	d = 1;
}

std::vector<Matrix>& FlattenLayer::FF(std::vector<Matrix>& _I) {
	double* o_ptr = O[0].d_data();
	auto sz = s_in.wh * sizeof(double);

	for (int i = 0; i < d_in; ++i) {
		_I[i].copyTo(I[i]);

		hipMemcpy(o_ptr + i*s_in.wh, I[i].d_data(), sz,
				hipMemcpyDeviceToDevice);
	}
	return O;
}

std::vector<Matrix>& FlattenLayer::BP(std::vector<Matrix>& _G) {
	double* g_ptr = _G[0].d_data();
	auto sz = s_in.wh * sizeof(double);

	for (int i = 0; i < d_in; ++i) {
		hipMemcpy(G[i].d_data(), g_ptr + i*s_in.wh, sz,
				hipMemcpyDeviceToDevice);
	}

	return G;
}

void FlattenLayer::update(){

}
