#include "hip/hip_runtime.h"
/*
 * Matrix.cpp
 *
 *  Created on: May 7, 2016
 *      Author: jamiecho
 */

#include "Matrix.h"
#include <functional>

using dfun = double (*)(double);

Matrix::Matrix(Size s, double* d)
:Matrix(s.n, s.m, d){
}
Matrix::Matrix(int n, int m, double* d) {
	int sz = n*m*sizeof(double);
	dat = (double*) malloc(sz);
	hipMalloc(&d_dat,sz);

	if(d != nullptr){
		hipMemcpy(d_dat,d,sz,hipMemcpyHostToDevice);
	}
}

Matrix::Matrix(Matrix& m){
	//copy constructor
	s = m.s;
	int sz = s.n * s.m * sizeof(double);

	hipMalloc(&d_dat, sz);
	hipMemcpy(d_dat,m.d_dat,sz,hipMemcpyDeviceToDevice);
}
Matrix::Matrix(Matrix&& m){
	//move constructor
	s = m.s;
	dat = m.dat;
	d_dat = m.d_dat;
	m.dat = nullptr;
	m.d_dat = nullptr;
}
Matrix::~Matrix() {
	free(dat);
	hipFree(d_dat);
	// TODO Auto-generated destructor stub
}

void Matrix::sync(){
	hipMemcpy(dat,d_dat,s.n*s.m*sizeof(double),hipMemcpyDeviceToHost);
}

__global__ void apply(double* I, dfun f){
	int i = threadIdx.x;
	I[i] = f(I[i]);

}

Matrix& Matrix::apply(dfun f){
	dfun f_d; //device function
	hipMemcpyFromSymbol(&f_d,HIP_SYMBOL(f),sizeof(dfun));
	apply(d_dat,f_d);
	//if 'device function' trick doesn't work, copy function to symbol with
	//hipMemcpyFromSymbol( &h_f[0], HIP_SYMBOL(pfunc1), sizeof(func));
	//or equivalent syntax.
	return *this;
}

void Matrix::zero(){
	hipMemset(d_dat,0,s.n*s.m*sizeof(double));
}

//Matrix& Matrix::apply(std::function<double(double)> f){
//
//	return *this;
//}
