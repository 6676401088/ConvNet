#include "hip/hip_runtime.h"
#include "PoolLayer.h"

__global__ void pool(double* I, double* O, int* SW, //Switch
		int iw, int ih, //width of input matrix
		int s_w, int s_h,  //stride dims
		int p_w, int p_h){ //pool dims

	//TODO: is 'max' pooling in terms of magnitude? or positive-max only?

	int h = blockDim.y;
	int w = blockDim.x;

	int i = threadIdx.y;
	int j = threadIdx.x;

	double maxVal = -99999.0;// reasonably small value, anyways.
	int maxIdx = 0;

	int index = idx(i,j,w);
	//TODO : fix all these arbitrary numbers

	for(int ii=0;ii<p_h && s_h*i+ii < ih;++ii){ //check i+ii for bounds
		for(int jj=0;jj<p_w && s_w*j+jj < iw;++jj){ //check j+jj for bounds

			int index_i = idx(s_h*i+ii,s_w*j+jj,iw);
			double val = I[index_i];

			if(val > maxVal){
				maxIdx = index_i; //switches, stored in flattened index
				maxVal = val;
			}
		}
	}

	O[index] = maxVal;
	SW[index] = maxIdx;
}

__global__ void invert_pool(double* G_o, double* G_i, int* SW){

	int i = threadIdx.x;
	G_i[SW[i]] = G_o[i];
}

PoolLayer::PoolLayer(Size s_s, Size s_p):s_s(s_s),s_p(s_p){

}

PoolLayer::~PoolLayer(){
	for(int i=0;i<d;++i){
		hipFree(SW[i]);
	}
	delete[] streams;
}

void PoolLayer::setup(Size& s, int& d){
	s_in = s;
	this->d = d;

	int w = s_in.w / s_s.w; //(s_in.w-s_p.w+s_s.w-1)/s_s.w;
	int h = s_in.h / s_s.h; //(s_in.h-s_p.h+s_s.h-1)/s_s.h;
	s_out = Size(w,h);

	SW.resize(d);

	streams = new hipStream_t[d];
	for(int i=0;i<d;++i){
		hipMalloc(&SW[i],sizeof(int) * w*h);
		//I.push_back(Matrix(s_in)); //doesn't need to allocate memory here
		G.push_back(Matrix(s_in));
		O.push_back(Matrix(s_out));
		hipStreamCreate(&streams[i]);

	}

	s = s_out;
	//no change for d
}

std::vector<Matrix>& PoolLayer::FF(std::vector<Matrix>& _I){
	dim3 blockDims(s_out.w, s_out.h);

	for(int i=0;i<d;++i){
		//_I[i].copyTo(I[i]);
		pool<<<1, blockDims>>>(_I[i].d_data(),O[i].d_data(),SW[i],
				s_in.w, s_in.h,
				s_s.w, s_s.h,
				s_p.w, s_p.h
				);
	}
	return O;
}


std::vector<Matrix>& PoolLayer::BP(std::vector<Matrix>& _G){
	for(int i=0;i<d;++i){
		//_G[i].set_sync(false);
		//namedPrint(_G[i]);
		G[i].zero();
		invert_pool<<<1,s_out.wh>>>(_G[i].d_data(),G[i].d_data(),SW[i]);
		//G[i].set_sync(false);
		//namedPrint(G[i]);

	}
	return G;
}

void PoolLayer::update(){

}
