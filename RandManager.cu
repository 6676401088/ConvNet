#include "RandManager.h"
#include <ctime>

RandManager::RandManager(){
    hiprandCreateGenerator (&rgen,HIPRAND_RNG_PSEUDO_DEFAULT );
    hiprandSetPseudoRandomGeneratorSeed (rgen ,time(0));
}

RandManager::~RandManager(){

}

void RandManager::rand(double* arr, int n){
	//TODO : get rid of this.
	hiprandGenerateUniformDouble(rgen,arr,n);
	//hiprandGenerateNormalDouble(rgen,arr,n,0.0,1.0);
}
void RandManager::randu(double* arr, int n){
	hiprandGenerateUniformDouble(rgen,arr,n);
}

void RandManager::randn(double* arr, int n, double mean=0.0, double stddev=1.0){
	hiprandGenerateNormalDouble(rgen,arr,n,mean,stddev);
}
