#include "hip/hip_runtime.h"
#include "RandManager.h"
#include <ctime>

__global__ void setup_rand( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void rand_gen(double* a, hiprandState* globalState)
{
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    a[ind] = RANDOM;
    globalState[ind] = localState;
}

RandManager::RandManager(int N):N(N){
	hipMalloc(&s, N*sizeof(hiprandState));
	setup_rand<<<1,N>>>(s,time(0));
}

RandManager::~RandManager(){
	hipFree(s);
}

void RandManager::rand(double* arr, int n){

	while (n > N) {
		rand_gen<<<1, N>>>(arr, s);
		arr += N;
		n -= N;
	}

	rand_gen<<<1,n>>>(arr,s);

}
