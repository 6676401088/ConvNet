#include "RandManager.h"
#include <ctime>

RandManager::RandManager(){
    hiprandCreateGenerator (&rgen,HIPRAND_RNG_PSEUDO_DEFAULT );
    hiprandSetPseudoRandomGeneratorSeed (rgen ,time(0));
}

RandManager::~RandManager(){

}

void RandManager::rand(double* arr, int n){
	hiprandGenerateUniformDouble(rgen,arr,n);
}
