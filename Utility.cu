#include "hip/hip_runtime.h"
#include "Utility.h"
#include <cassert>
#include <string>

/* n < 1024 */
__global__ void _add(const double* a, const double* b, double* out) {
	int i = threadIdx.x;
	out[i] = a[i] + b[i];
}
__global__ void _sub(const double* a, const double* b, double* out) {
	int i = threadIdx.x;
	out[i] = a[i] - b[i];
}
__global__ void _mul(const double* a, const double* b, double* out) {
	int i = threadIdx.x;
	out[i] = a[i] * b[i];
}
__global__ void _div(const double* a, const double* b, double* out) {
	int i = threadIdx.x;
	out[i] = a[i] / b[i];
}

__global__ void _add(const double* a, const double b, double* out) {
	int i = threadIdx.x;
	out[i] = a[i] + b;
}
__global__ void _sub(const double* a, const double b, double* out) {
	int i = threadIdx.x;
	out[i] = a[i] - b;
}
__global__ void _mul(const double* a, const double b, double* out) {
	int i = threadIdx.x;
	out[i] = a[i] * b;
}
__global__ void _div(const double* a, const double b, double* out) {
	int i = threadIdx.x;
	out[i] = a[i] / b;
}

__global__ void _abs(const double* in, double* out) { //what if in == out? well...
	int i = threadIdx.x;
	out[i] = in[i] > 0 ? in[i] : -in[i];
}

/* n >= 1024 */
__global__ void _add(const double* a, const double* b, double* out, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = a[i] + b[i];
}
__global__ void _sub(const double* a, const double* b, double* out, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = a[i] - b[i];
}
__global__ void _mul(const double* a, const double* b, double* out, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = a[i] * b[i];
}
__global__ void _div(const double* a, const double* b, double* out, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = a[i] / b[i];
}

__global__ void _add(const double* a, const double b, double* out, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = a[i] + b;
}
__global__ void _sub(const double* a, const double b, double* out, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = a[i] - b;
}
__global__ void _mul(const double* a, const double b, double* out, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = a[i] * b;
}
__global__ void _div(const double* a, const double b, double* out, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		out[i] = a[i] / b;
}

void add(const double* a, const double* b, double* o, int n) {
	if (n < 1024) {
		_add<<<1,n>>>(a,b,o);
	} else {
		int nb = (n + 255) / 256; //# of blocks
		_add<<<nb,256>>>(a,b,o,n);
	}
}
void sub(const double* a, const double* b, double* o, int n) {
	if (n < 1024) {
		_sub<<<1,n>>>(a,b,o);
	} else {
		int nb = (n + 255) / 256; //# of blocks
		_sub<<<nb,256>>>(a,b,o,n);
	}
}
void mul(const double* a, const double* b, double* o, int n) {
	if (n < 1024) {
		_mul<<<1,n>>>(a,b,o);
	} else {
		int nb = (n + 255) / 256; //# of blocks
		_mul<<<nb,256>>>(a,b,o,n);
	}
}
void div(const double* a, const double* b, double* o, int n) {
	if (n < 1024) {
		_div<<<1,n>>>(a,b,o);
	} else {
		int nb = (n + 255) / 256; //# of blocks
		_div<<<nb,256>>>(a,b,o,n);
	}
}

void add(const double* a, const double b, double* o, int n) {
	if (n < 1024) {
		_add<<<1,n>>>(a,b,o);
	} else {
		int nb = (n + 255) / 256; //# of blocks
		_add<<<nb,256>>>(a,b,o,n);
	}
}
void sub(const double* a, const double b, double* o, int n) {
	if (n < 1024) {
		_sub<<<1,n>>>(a,b,o);
	} else {
		int nb = (n + 255) / 256; //# of blocks
		_sub<<<nb,256>>>(a,b,o,n);
	}
}
void mul(const double* a, const double b, double* o, int n) {
	if (n < 1024) {
		_mul<<<1,n>>>(a,b,o);
	} else {
		int nb = (n + 255) / 256; //# of blocks
		_mul<<<nb,256>>>(a,b,o,n);
	}
}
void div(const double* a, const double b, double* o, int n) {
	if (n < 1024) {
		_div<<<1,n>>>(a,b,o);
	} else {
		int nb = (n + 255) / 256; //# of blocks
		_div<<<nb,256>>>(a,b,o,n);
	}
}

void abs(const double* in, double* out, int n) {
	//TODO : reimplement for robustness
	_abs<<<1,n>>>(in,out);
}


__global__ void _convolve(const double* d_i, const double* d_k, double* d_o,
		int r) {
	int i = threadIdx.y;
	int j = threadIdx.x;

	int h = blockDim.y;
	int w = blockDim.x;

	d_o[idx(i, j, w)] = 0;
	for (int ki = -r; ki <= r; ++ki) {
		for (int kj = -r; kj <= r; ++kj) {
			if (inbound(i + ki, j + kj, h, w)) {
				d_o[idx(i, j, w)] += d_i[idx(i + ki, j + kj, w)]
						* d_k[idx(r - ki, r - kj, 2 * r + 1)]; //flip here if correlation
			}
			//effectively zero-padding
			//may change to VALID convolution later

			//d_o[i][j] += d_i[i+ki][j+kj] * d_k[ki+r][kj+r]
		}
	}

}
__global__ void _correlate(const double* d_i, const double* d_k, double* d_o,
		int r) {
	int i = threadIdx.y;
	int j = threadIdx.x;

	int h = blockDim.y;
	int w = blockDim.x;

	d_o[idx(i, j, w)] = 0;
	for (int ki = -r; ki <= r; ++ki) {
		for (int kj = -r; kj <= r; ++kj) {
			if (inbound(i + ki, j + kj, h, w)) {
				d_o[idx(i, j, w)] += d_i[idx(i + ki, j + kj, w)]
						* d_k[idx(r + ki, r + kj, 2 * r + 1)]; //flipped here, for correlation
			}
			//effectively zero-padding
			//may change to VALID convolution later

			//d_o[i][j] += d_i[i+ki][j+kj] * d_k[ki+r][kj+r]
		}
	}
}
void convolve_d(const double* d_i, const double* d_k, double* d_o,
//if all ptrs are in gpu
		int w, int h, int r, hipStream_t* stream) {
	dim3 g(1, 1);
	dim3 b(w, h);
	if (stream) {
		_convolve<<<g,b,0,*stream>>>(d_i,d_k,d_o,r);
	} else {
		_convolve<<<g,b>>>(d_i,d_k,d_o,r);
	}

}

void correlate_d(const double* d_i, const double* d_k, double* d_o,
//if all ptrs are in gpu
		int w, int h, int r, hipStream_t* stream) {
	dim3 g(1, 1);
	dim3 b(w, h);
	if (stream) {
		_correlate<<<g,b,0,*stream>>>(d_i,d_k,d_o,r);
	} else {
		_correlate<<<g,b>>>(d_i,d_k,d_o,r);
	}
}

void convolve(const double* i, const double* k, double* o, int w, int h,
		int r) {

	double* d_i, *d_k, *d_o;

	int sz = w * h * sizeof(double);
	int ksz = (2 * r + 1) * (2 * r + 1) * sizeof(double);

	hipMalloc(&d_i, sz);
	hipMalloc(&d_k, ksz);
	hipMalloc(&d_o, sz);

	hipMemcpy(d_i, i, sz, hipMemcpyHostToDevice);
	hipMemcpy(d_k, k, ksz, hipMemcpyHostToDevice);

	//clock_t start = clock();
	convolve_d(d_i, d_k, d_o, w, h, r);

	//clock_t end = clock();
	//printf("Took %f Seconds", float(end-start)/CLOCKS_PER_SEC);

	hipMemcpy(o, d_o, sz, hipMemcpyDeviceToHost);

	hipFree(d_i);
	hipFree(d_k);
	hipFree(d_o);

	return;
}

__global__ void gridMax(double* arr, int n, double* b_max) { //b_sum = block-sum
	extern __shared__ double s_arr[]; //blockDim.x;

	int start = blockIdx.x * blockDim.x;
	int i = start + threadIdx.x;
	int ti = threadIdx.x;

	if (i >= n) //o.o.b
		return;

	s_arr[ti] = arr[i]; // copy to shared memory
	__syncthreads(); //guarantee complete copy

	int nt = NearestPowerOf2(blockDim.x); //num threads in block
	//reduction within block...
	for (int half = (nt >> 1); half > 0; half >>= 1) {
		if (ti < half) {
			int ti_2 = ti + half;
			if (start + ti_2 < n) { //within bounds
				s_arr[ti] = max(s_arr[ti], s_arr[ti_2]);
			}
		}
		__syncthreads();
	}
	__syncthreads();

	if (ti == 0) { // 1 per block
		b_max[blockIdx.x] = s_arr[0];
	}
}

__device__ int NearestPowerOf2 (int n)
{
  if (!n) return n;  //(0 == 2^0)

  int x = 1;
  while(x < n)
    {
      x <<= 1;
    }
  return x;
}

__global__ void gridMin(double* arr, int n, double* b_min) { //b_sum = block-sum
	extern __shared__ double s_arr[]; //blockDim.x;

	int start = blockIdx.x * blockDim.x;
	int i = start + threadIdx.x;
	int ti = threadIdx.x;

	if (i >= n) //o.o.b
		return;

	s_arr[ti] = arr[i]; // copy to shared memory
	__syncthreads(); //guarantee complete copy

	int nt = NearestPowerOf2(blockDim.x); //num threads in block
	//reduction within block...
	for (int half = (nt >> 1); half > 0; half >>= 1) {
		if (ti < half) {
			int ti_2 = ti + half;
			if (start + ti_2 < n) { //within bounds
				s_arr[ti] = min(s_arr[ti], s_arr[ti_2]);
			}
		}
		__syncthreads();
	}
	__syncthreads();

	if (ti == 0) { // 1 per block
		b_min[blockIdx.x] = s_arr[0];
	}
}

__global__ void gridSum(double* arr, int n, double* b_sum) { //b_sum = block-sum
	extern __shared__ double s_arr[]; //blockDim.x;
	int start = blockIdx.x * blockDim.x;
	int i = start + threadIdx.x;
	int ti = threadIdx.x;

	if (i >= n) //o.o.b
		return;

	s_arr[ti] = arr[i]; // copy to shared memory
	__syncthreads(); //guarantee complete copy

	int nt = NearestPowerOf2(blockDim.x); //num threads in block
	//reduction within block...
	for (int half = (nt >> 1); half > 0; half >>= 1) {
		if (ti < half) {
			int ti_2 = ti + half;
			if (start + ti_2 < n) { //within bounds
				s_arr[ti] += s_arr[ti_2];
			}
		}
		__syncthreads();
	}
	__syncthreads();

	if (ti == 0) { // 1 per block
		b_sum[blockIdx.x] = s_arr[0];
	}
}

double reduce(double* d_arr, int n, std::string op) {
	assert(n < 65536);

	double* d_tmp, *d_res;
	double res = 0;

	dim3 gridDims((n + 255) / 256);
	dim3 blockDims(256);

	hipMalloc(&d_tmp, sizeof(double) * gridDims.x);
	hipMalloc(&d_res, sizeof(double));

	if(op == "sum"){
		gridSum<<<gridDims,blockDims,sizeof(double)*256>>>(d_arr,n,d_tmp);
		gridSum<<<1,gridDims,sizeof(double)>>>(d_tmp,gridDims.x,d_res);
	}else if(op == "min"){
		gridMin<<<gridDims,blockDims,sizeof(double)*256>>>(d_arr,n,d_tmp);
		gridMin<<<1,gridDims,sizeof(double)>>>(d_tmp,gridDims.x,d_res);
	}else if(op == "max"){
		gridMax<<<gridDims,blockDims,sizeof(double)*256>>>(d_arr,n,d_tmp);
		gridMax<<<1,gridDims,sizeof(double)>>>(d_tmp,gridDims.x,d_res);
	}

	hipMemcpy(&res, d_res, sizeof(double), hipMemcpyDeviceToHost);

	return res;
}
